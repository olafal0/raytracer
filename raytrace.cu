#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cfloat>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "imshow.h"
#include "vecmath.h"

// for GTX 760:
// 1024 threads per block
// 2048 threads per SM
// 16 blocks per SM
// 6 SMs

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCKS 16*6
// use WARPS_PER_BLOCK to set blockDim.y, warpSize to set blockDim.x
#define WARPS_PER_BLOCK (THREADS_PER_BLOCK/32)

// thanks go to Lode Vandevenne for the LodePNG examples

// each warp (each 32 threads) is reponsible for one pixel
// pixel id = threadIdx.y + blockIdx.x * blockDim.x
// threadIdx.x is id (0..31) of thread in the warp for that pixel
// each thread should perform a warp-stride loop over sphere list and find its best match
// then, log2(32)=5 loops of warp shuffles to reduce values,
// until threadIdx.x=0 has the closest hit sphere

// kernel function should take a pixel, construct a ray for it, and cast against all spheres in the scene
__global__
void getColorAtPixel (int w, int h, float fovtan, float fovtanAspect, v3 origin, unsigned char *rgba, float *sphereList, int numSpheres) {
  // copy the whole sphere list into shared memory
  // this requires 16KiB per block for 1000 spheres, unlimited spheres are not yet handled
  extern __shared__ float spheres[];
  for (int i=threadIdx.x+(threadIdx.y*blockDim.x);i<numSpheres*4;i+=blockDim.x*blockDim.y) {
    spheres[i] = sphereList[i];
  }
  __syncthreads();
  float *px = &spheres[0];
  float *py = &spheres[numSpheres];
  float *pz = &spheres[numSpheres*2];
  float *rad = &spheres[numSpheres*3];
  
  // do for each pixel...
  int idx = (threadIdx.y + blockIdx.x * blockDim.y);
  for (; idx < w*h; idx += blockDim.y*gridDim.x) {

    // get pixel coords
    int x = idx % w;
    int y = idx / w;

    // find out which thread in the warp we are
    int wid = threadIdx.x;

    // get normalized ray direction
    float dirx, diry, dirz;
    dirx = (2*x-(float)w) / ((float)w) * fovtan;
    diry = ((float)h-2*y) / ((float)h) * fovtanAspect;
    dirz = 1;
    float magn = sqrtf(dirx*dirx + diry*diry + dirz*dirz);
    dirx /= magn;
    diry /= magn;
    dirz /= magn;

    float origx, origy, origz;
    origx = origin.x;
    origy = origin.y;
    origz = origin.z;

    float distanceX, distanceY, distanceZ, dotProduct, distanceSqr, importantPart, d;

    float bestDist =  FLT_MAX;
    int bestHitSphere = -1;
    v3 bestPt, bestNorm;

    for (int i=wid; i<numSpheres; i+=blockDim.x) {
      distanceX = origx - px[i];
      distanceY = origy - py[i];
      distanceZ = origz - pz[i];

      dotProduct = dirx*(distanceX) + diry*(distanceY) + dirz*(distanceZ);
      distanceSqr = distanceX*distanceX + distanceY*distanceY + distanceZ*distanceZ;
      importantPart = dotProduct*dotProduct - distanceSqr + rad[i]*rad[i];

      d = -dotProduct - sqrtf(importantPart);
      if (d < bestDist) {
        bestDist = d;
        bestHitSphere = i;
      }
    }

    // reduce to minimum using warp shuffles
    for (int d=1; d<warpSize; d*=2) {
      float otherDist = __shfl_down(bestDist,d);
      float otherBest = __shfl_down(bestHitSphere,d);
      if (otherDist < bestDist) {
        bestDist = otherDist;
        bestHitSphere = otherBest;
      }
    }

    int pixidx = idx*4;

    // use other threads in warp for writing
    bestDist = __shfl(bestDist,0);
    bestHitSphere = __shfl(bestHitSphere,0);
    unsigned char pixvalue = 0;
    if (bestHitSphere >= 0) {
      bestPt.x = origx + dirx*bestDist;
      bestPt.y = origy + diry*bestDist;
      bestPt.z = origz + dirz*bestDist;
      //bestNorm.y = (py[i] - bestPt.y) * (1.0/rad[i]);
      //bestNorm.x = (px[i] - bestPt.x) * (1.0/rad[i]);
      bestNorm.z = (pz[bestHitSphere] - bestPt.z) * (1.0/rad[bestHitSphere]);
      float normalDot = bestNorm.z;
      if (normalDot<0) normalDot = 0;
      pixvalue = (normalDot)*255;
    }
    if (wid==3) pixvalue = 255;
    if (wid < 4) {
      rgba[pixidx+wid] = pixvalue;
    }
  }
  // ops: 28 per pixel + (19 per sphere)
}

void errorCheck (int errorCode) {
  if (errorCode != hipSuccess) {
    printf("%s\n", hipGetErrorString(hipGetLastError()));
  }
  return;
}

int main(int argc, char* argv[]) {
  if (argc < 2) {
    std::cout << "Usage: raytrace numSpheres [show]\n";
    return 0;
  }
  int nSpheres = atoi(argv[1]);
  int iterations;
  if (argc > 2) iterations = 1;
  else iterations = 10;
  // make a 512x512 image
  uint w, h;
  w = 1920;
  h = 1080;
  const int sz = w*h;
  unsigned char *rgba = new unsigned char[sz*4];

  // we only need to allocate device's texture memory, the GPU will set it
  unsigned char *drgba;
  errorCheck(hipMalloc(&drgba,sizeof(unsigned char)*sz*4));

  // construct the camera/screen
  view cam = view(w,h,90.0);
  cam.pos = vec3(0,0,-5);
  cam.fwd = vec3(0,0,1); // straight forward

  // allocate host spheres as one array
  float *spheres, *px, *py, *pz, *rad;
  spheres = (float*)calloc(nSpheres*4,sizeof(float));
  px = &(spheres[0]);
  py = &spheres[nSpheres];
  pz = &spheres[nSpheres*2];
  rad = &spheres[nSpheres*3];

  // allocate device spheres
  //float *dpx, *dpy, *dpz, *drad;
  // errorCheck(hipMalloc(&dpx,sizeof(float)*nSpheres));
  // errorCheck(hipMalloc(&dpy,sizeof(float)*nSpheres));
  // errorCheck(hipMalloc(&dpz,sizeof(float)*nSpheres));
  // errorCheck(hipMalloc(&drad,sizeof(float)*nSpheres));
  float *dspheres;
  errorCheck(hipMalloc(&dspheres,sizeof(float)*nSpheres*4));

  // initialize host spheres
  for (int i=0; i<nSpheres; i++) {
    px[i] = ((i%2)*2-1)*0.05*i;
    py[i] = (((i/2)%2)*2-1)*0.05*i;
    pz[i] = i*0.1;
    rad[i]= 0.5;
  }

  // copy host spheres to device
  // errorCheck(hipMemcpy(dpx,px,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  // errorCheck(hipMemcpy(dpy,py,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  // errorCheck(hipMemcpy(dpz,pz,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  // errorCheck(hipMemcpy(drad,rad,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  errorCheck(hipMemcpy(dspheres,spheres,sizeof(float)*nSpheres*4,hipMemcpyHostToDevice));
  hipDeviceSynchronize();

  std::chrono::time_point<std::chrono::system_clock> start, end;
  std::chrono::duration<double> elapsed_seconds;

  float fvtan = cam.fovtan;
  float fvtanAsp = cam.fovtanAspect;
  v3 orig;
  orig.x = cam.pos.x;
  orig.y = cam.pos.y;
  orig.z = cam.pos.z;

  //hipDeviceSetCacheConfig(hipFuncCachePreferL1);
  dim3 blockSize(32,WARPS_PER_BLOCK);

  start = std::chrono::system_clock::now();

  for (int iter=0; iter<iterations; iter++) {
    getColorAtPixel<<<NUM_BLOCKS, blockSize, nSpheres*4*sizeof(float)>>>(w,h,fvtan,fvtanAsp,orig,drgba,dspheres,nSpheres);
  }
  hipDeviceSynchronize();
  errorCheck(hipGetLastError());

  end = std::chrono::system_clock::now();
  elapsed_seconds = end-start;
  std::cout << (elapsed_seconds.count()*1000.0/iterations) << "\n";

  hipMemcpy(rgba,drgba,sizeof(unsigned char)*sz*4,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  if (argc > 2) show("Sample image", rgba, w, h);

  free(spheres);
  delete[] rgba;
  hipFree(dspheres);
  hipFree(drgba);
}