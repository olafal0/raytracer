#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cfloat>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "imshow.h"
#include "vecmath.h"

#define THREADS_PER_BLOCK 1024
#define NUM_BLOCKS 16*6
#define CHUNK_SIZE (THREADS_PER_BLOCK*NUM_BLOCKS)

// thanks go to Lode Vandevenne for the LodePNG library and examples

// kernel function should take a pixel, construct a ray for it, and cast against all spheres in the scene
__global__
void getColorAtPixel (int startIdx, int w, int h, float fovtan, float fovtanAspect, v3 origin, unsigned char *rgba, float *sphereList, int numSpheres) {
  // copy the whole sphere list into shared memory
  // this requires 16KiB per block for 1000 spheres, and more than 1000 spheres is not yet handled
  extern __shared__ float spheres[];
  for (int i=threadIdx.x;i<numSpheres*4;i+=blockDim.x) {
    spheres[i] = sphereList[i];
  }
  __syncthreads();
  float *px = &spheres[0];
  float *py = &spheres[numSpheres];
  float *pz = &spheres[numSpheres*2];
  float *rad = &spheres[numSpheres*3];

  int idx = (threadIdx.x + blockIdx.x * blockDim.x) + startIdx;
  for (idx; idx < w*h; idx += blockDim.x*gridDim.x) {

    // get pixel coords
    int x = idx % w;
    int y = idx / w;

    // get normalized ray direction
    float dirx, diry, dirz;
    dirx = (2*x-(float)w) / ((float)w) * fovtan;
    diry = ((float)h-2*y) / ((float)h) * fovtanAspect;
    dirz = 1;
    float magn = sqrtf(dirx*dirx + diry*diry + dirz*dirz);
    dirx /= magn;
    diry /= magn;
    dirz /= magn;

    float origx, origy, origz;
    origx = origin.x;
    origy = origin.y;
    origz = origin.z;

    float distanceX, distanceY, distanceZ, dotProduct, distanceSqr, importantPart, d;

    float bestDist =  FLT_MAX;
    int bestHitSphere = -1;
    v3 bestPt, bestNorm;

    /*
      Accesses for this thread:
      p{x,y,z}[0..n]
      rad[0..n]
      then write to rgba[idx*4..idx*4+4]
    */

    for (int i=0; i<numSpheres; i++) {
      distanceX = origx - px[i];
      distanceY = origy - py[i];
      distanceZ = origz - pz[i];

      dotProduct = dirx*(distanceX) + diry*(distanceY) + dirz*(distanceZ);
      distanceSqr = distanceX*distanceX + distanceY*distanceY + distanceZ*distanceZ;
      importantPart = dotProduct*dotProduct - distanceSqr + rad[i]*rad[i];

      d = -dotProduct - sqrtf(importantPart);
      if (d < bestDist) {
        bestDist = d;
        bestHitSphere = i;
      }
    }


    unsigned char pixvalue = 0;
    if (bestHitSphere >= 0) {
      bestPt.x = origx + dirx*bestDist;
      bestPt.y = origy + diry*bestDist;
      bestPt.z = origz + dirz*bestDist;
      //bestNorm.y = (py[i] - bestPt.y) * (1.0/rad[i]);
      //bestNorm.x = (px[i] - bestPt.x) * (1.0/rad[i]);
      bestNorm.z = (pz[bestHitSphere] - bestPt.z) * (1.0/rad[bestHitSphere]);
      float normalDot = bestNorm.z;
      if (normalDot<0) normalDot = 0;
      pixvalue = (normalDot)*255;
    }
    int pixidx = idx*4;
    rgba[pixidx+0] = pixvalue;
    rgba[pixidx+1] = pixvalue;
    rgba[pixidx+2] = pixvalue;
    rgba[pixidx+3] = 255;
  }
}

void errorCheck (int errorCode) {
  if (errorCode != hipSuccess) {
    printf("%s\n", hipGetErrorString(hipGetLastError()));
  }
  return;
}

int main(int argc, char* argv[]) {
  if (argc < 2) {
    std::cout << "Usage: raytrace numSpheres [show]\n";
    return 0;
  }
  int nSpheres = atoi(argv[1]);
  int iterations;
  if (argc > 2) iterations = 1;
  else iterations = 100;
  // make a 512x512 image
  uint w, h;
  w = 1920;
  h = 1080;
  const int sz = w*h;
  unsigned char *rgba = new unsigned char[sz*4];

  // we only need to allocate device's texture memory, the GPU will set it
  unsigned char *drgba;
  errorCheck(hipMalloc(&drgba,sizeof(unsigned char)*sz*4));

  // construct the camera/screen
  view cam = view(w,h,90.0);
  cam.pos = vec3(0,0,-5);
  cam.fwd = vec3(0,0,1); // straight forward

  // allocate host spheres
  float *px, *py, *pz, *rad, *spheres;
  spheres = (float*)calloc(nSpheres*4,sizeof(float));
  // px = (float*)calloc(nSpheres,sizeof(float));
  // py = (float*)calloc(nSpheres,sizeof(float));
  // pz = (float*)calloc(nSpheres,sizeof(float));
  // rad = (float*)calloc(nSpheres,sizeof(float));
  px = &(spheres[0]);
  py = &spheres[nSpheres];
  pz = &spheres[nSpheres*2];
  rad = &spheres[nSpheres*3];

  // allocate device spheres
  //float *dpx, *dpy, *dpz, *drad;
  // errorCheck(hipMalloc(&dpx,sizeof(float)*nSpheres));
  // errorCheck(hipMalloc(&dpy,sizeof(float)*nSpheres));
  // errorCheck(hipMalloc(&dpz,sizeof(float)*nSpheres));
  // errorCheck(hipMalloc(&drad,sizeof(float)*nSpheres));
  float *dspheres;
  errorCheck(hipMalloc(&dspheres,sizeof(float)*nSpheres*4));

  // initialize host spheres
  for (int i=0; i<nSpheres; i++) {
    px[i] = ((i%2)*2-1)*0.05*i;
    py[i] = (((i/2)%2)*2-1)*0.05*i;
    pz[i] = i*0.1;
    rad[i]= 0.5;
  }

  // copy host spheres to device
  // errorCheck(hipMemcpy(dpx,px,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  // errorCheck(hipMemcpy(dpy,py,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  // errorCheck(hipMemcpy(dpz,pz,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  // errorCheck(hipMemcpy(drad,rad,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  errorCheck(hipMemcpy(dspheres,spheres,sizeof(float)*nSpheres*4,hipMemcpyHostToDevice));
  hipDeviceSynchronize();

  std::chrono::time_point<std::chrono::system_clock> start, end;
  std::chrono::duration<double> elapsed_seconds;

  start = std::chrono::system_clock::now();

  //void getColorAtPixel (int startIdx, int w, int h, float fovtan, float fovtanAspect, vec3 origin, unsigned char *rgba, float *px, float *py, float *pz, float *rad, int numSpheres)

  float fvtan = cam.fovtan;
  float fvtanAsp = cam.fovtanAspect;
  v3 orig;
  orig.x = cam.pos.x;
  orig.y = cam.pos.y;
  orig.z = cam.pos.z;

  for (int iter=0; iter<iterations; iter++) {
    getColorAtPixel<<<NUM_BLOCKS, THREADS_PER_BLOCK, nSpheres*4*sizeof(float)>>>(0,w,h,fvtan,fvtanAsp,orig,drgba,dspheres,nSpheres);
    hipDeviceSynchronize();
    errorCheck(hipGetLastError());
  }

  end = std::chrono::system_clock::now();
  elapsed_seconds = end-start;
  std::cout << (elapsed_seconds.count()*1000.0/iterations) << "\n";

  hipMemcpy(rgba,drgba,sizeof(unsigned char)*sz*4,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  if (argc > 2) show("Sample image", rgba, w, h);

  free(spheres);
  delete[] rgba;
  hipFree(dspheres);
}