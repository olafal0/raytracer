#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cfloat>
#include <omp.h>
#include <hip/hip_runtime.h>
#include "imshow.h"
#include "vecmath.h"

#define THREADS_PER_BLOCK 512
#define NUM_BLOCKS 512
#define CHUNK_SIZE (THREADS_PER_BLOCK*NUM_BLOCKS)

// thanks go to Lode Vandevenne for the LodePNG library and examples

// kernel function should take a pixel, construct a ray for it, and cast against all spheres in the scene
__global__
void getColorAtPixel (int startIdx, int w, int h, float fovtan, float fovtanAspect, v3 origin, unsigned char *rgba, float *px, float *py, float *pz, float *rad, int numSpheres) {
  int idx = (threadIdx.x + blockIdx.x * blockDim.x) + startIdx;
  if (idx > w*h) return;

  // get pixel coords
  int x = idx % w;
  int y = idx / w;

  // get normalized ray direction
  float dirx, diry, dirz;
  dirx = (2*x-(float)w) / ((float)w) * fovtan;
  diry = ((float)h-2*y) / ((float)h) * fovtanAspect;
  dirz = 1;
  float magn = sqrtf(dirx*dirx + diry*diry + dirz*dirz);
  dirx /= magn;
  diry /= magn;
  dirz /= magn;

  float origx, origy, origz;
  origx = origin.x;
  origy = origin.y;
  origz = origin.z;

  float distanceX, distanceY, distanceZ, dotProduct, distanceSqr, importantPart, d;

  float bestDist =  FLT_MAX;
  v3 bestPt, bestNorm;
  bool gotBestHit = false;

  for (int i=0; i<numSpheres; i++) {
    distanceX = origx - px[i];
    distanceY = origy - py[i];
    distanceZ = origz - pz[i];

    dotProduct = dirx*(distanceX) + diry*(distanceY) + dirz*(distanceZ);
    distanceSqr = distanceX*distanceX + distanceY*distanceY + distanceZ*distanceZ;
    importantPart = dotProduct*dotProduct - distanceSqr + rad[i]*rad[i];

    d = -dotProduct - sqrtf(importantPart);
    if (d < bestDist) {
      bestPt.x = origx + dirx*d;
      bestPt.y = origy + diry*d;
      bestPt.z = origz + dirz*d;
      //bestNorm.y = (py[i] - bestPt.y) * (1.0/rad[i]);
      //bestNorm.x = (px[i] - bestPt.x) * (1.0/rad[i]);
      bestNorm.z = (pz[i] - bestPt.z) * (1.0/rad[i]);
      bestDist = d;

      gotBestHit = true;
    }
  }


  unsigned char pixvalue = 0;
  if (gotBestHit) {
    float normalDot = bestNorm.z;
    if (normalDot<0) normalDot = 0;
    pixvalue = (normalDot)*255;
  }
  int pixidx = idx*4;
  rgba[pixidx+0] = pixvalue;
  rgba[pixidx+1] = pixvalue;
  rgba[pixidx+2] = pixvalue;
  rgba[pixidx+3] = 255;
}

void errorCheck (int errorCode) {
  if (errorCode != hipSuccess) {
    printf("%s\n", hipGetErrorString(hipGetLastError()));
  }
  return;
}

int main(int argc, char* argv[]) {
  if (argc < 2) {
    std::cout << "Usage: raytrace numSpheres [show]\n";
    return 0;
  }
  int nSpheres = atoi(argv[1]);
  int iterations;
  if (argc > 2) iterations = 1;
  else iterations = 100;
  // make a 512x512 image
  uint w, h;
  w = 1920;
  h = 1080;
  const int sz = w*h;
  unsigned char *rgba = new unsigned char[sz*4];

  // we only need to allocate device's texture memory, the GPU will set it
  unsigned char *drgba;
  errorCheck(hipMalloc(&drgba,sizeof(unsigned char)*sz*4));

  // construct the camera/screen
  view cam = view(w,h,90.0);
  cam.pos = vec3(0,0,-5);
  cam.fwd = vec3(0,0,1); // straight forward

  // allocate host spheres
  float *px, *py, *pz, *rad;
  px = (float*)calloc(nSpheres,sizeof(float));
  py = (float*)calloc(nSpheres,sizeof(float));
  pz = (float*)calloc(nSpheres,sizeof(float));
  rad = (float*)calloc(nSpheres,sizeof(float));

  // allocate device spheres
  float *dpx, *dpy, *dpz, *drad;
  errorCheck(hipMalloc(&dpx,sizeof(float)*nSpheres));
  errorCheck(hipMalloc(&dpy,sizeof(float)*nSpheres));
  errorCheck(hipMalloc(&dpz,sizeof(float)*nSpheres));
  errorCheck(hipMalloc(&drad,sizeof(float)*nSpheres));

  // initialize host spheres
  for (int i=0; i<nSpheres; i++) {
    px[i] = ((i%2)*2-1)*0.05*i;
    py[i] = (((i/2)%2)*2-1)*0.05*i;
    pz[i] = i*0.1;
    rad[i]= 0.5;
  }

  // copy host spheres to device
  errorCheck(hipMemcpy(dpx,px,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  errorCheck(hipMemcpy(dpy,py,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  errorCheck(hipMemcpy(dpz,pz,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  errorCheck(hipMemcpy(drad,rad,sizeof(float)*nSpheres,hipMemcpyHostToDevice));
  hipDeviceSynchronize();

  std::chrono::time_point<std::chrono::system_clock> start, end;
  std::chrono::duration<double> elapsed_seconds;

  start = std::chrono::system_clock::now();

  //void getColorAtPixel (int startIdx, int w, int h, float fovtan, float fovtanAspect, vec3 origin, unsigned char *rgba, float *px, float *py, float *pz, float *rad, int numSpheres)

  float fvtan = cam.fovtan;
  float fvtanAsp = cam.fovtanAspect;
  v3 orig;
  orig.x = cam.pos.x;
  orig.y = cam.pos.y;
  orig.z = cam.pos.z;

  for (int iter=0; iter<iterations; iter++) {
    for (int i=0; i<sz; i+=CHUNK_SIZE) {
      getColorAtPixel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(i,w,h,fvtan,fvtanAsp,orig,drgba,dpx,dpy,dpz,drad,nSpheres);
    }
    hipDeviceSynchronize();
    errorCheck(hipGetLastError());
    
  }

  end = std::chrono::system_clock::now();
  elapsed_seconds = end-start;
  std::cout << (elapsed_seconds.count()*1000.0/iterations) << "\n";

  hipMemcpy(rgba,drgba,sizeof(unsigned char)*sz*4,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  if (argc > 2) show("Sample image", rgba, w, h);

  free(px);
  free(py);
  free(pz);
  free(rad);
  delete[] rgba;
}